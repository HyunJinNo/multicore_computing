#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <chrono>
#include <iostream>
#include "hip/hip_runtime.h"


using namespace std;
using namespace std::chrono;

#define SPHERES 20
#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
	float r, b, g;
	float radius;
	float x, y, z;
	float hit(float ox, float oy, float* n) {
		float dx = ox - x;
		float dy = oy - y;
		if (dx * dx + dy * dy < radius * radius) {
			float dz = sqrtf(radius * radius - dx * dx - dy * dy);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}
		return -INF;
	}
};

__global__ void kernel(Sphere* s, unsigned char* ptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * DIM;
	float ox = (x - DIM / 2);
	float oy = (y - DIM / 2);

	//printf("x:%d, y:%d, ox:%f, oy:%f\n", x, y, ox, oy);

	int i;
	float r = 0, g = 0, b = 0;
	float maxz = -INF;

	for (i = 0; i < SPHERES; i++) {
		float n;
		//float t = s[i].hit(ox, oy, &n);
		float t;
		float dx = ox - s[i].x;
		float dy = oy - s[i].y;
		float radius = s[i].radius;
		if (dx * dx + dy * dy < radius * radius) {
			float dz = sqrtf(radius * radius - dx * dx - dy * dy);
			n = dz / sqrtf(radius * radius);
			t = dz + s[i].z;
		}
		else {
			t = -INF;
		}
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		}
	}

	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp) {
	int i, x, y;
	fprintf(fp, "P3\n");
	fprintf(fp, "%d %d\n", xdim, ydim);
	fprintf(fp, "255\n");

	for (y = 0; y < ydim; y++) {
		for (x = 0; x < xdim; x++) {
			i = x + y * xdim;
			fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
		}
		fprintf(fp, "\n");
	}
}

int main() {
	int i;
	Sphere* temp_s;
	Sphere* d_temp_s;
	unsigned char* bitmap;
	unsigned char* d_bitmap;
	int size1 = sizeof(Sphere) * SPHERES;
	int size2 = sizeof(unsigned char) * DIM * DIM * 4;
	FILE* fp = fopen("result.ppm", "w");
	srand(time(NULL));

	// Allocate space for device copies of temp_s and bitmap
	hipMalloc((void**)&d_temp_s, size1);
	hipMalloc((void**)&d_bitmap, size2);

	// Allocate space for host copies of temp_s and bitmap
	temp_s = (Sphere*)malloc(size1);
	bitmap = (unsigned char*)malloc(size2);

	// Setup initial values
	for (i = 0; i < SPHERES; i++) {
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(2000.0f) - 1000;
		temp_s[i].y = rnd(2000.0f) - 1000;
		temp_s[i].z = rnd(2000.0f) - 1000;
		temp_s[i].radius = rnd(200.0f) + 40;
	}

	auto start_time = high_resolution_clock::now();

	// Copy values to device
	hipMemcpy(d_temp_s, temp_s, size1, hipMemcpyHostToDevice);

	// Setup the execution configuration
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid(64, 64, 1);

	kernel<<<dimGrid, dimBlock>>>(d_temp_s, d_bitmap);

	// Copy result back to host
	hipMemcpy(bitmap, d_bitmap, size2, hipMemcpyDeviceToHost);

	auto end_time = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(end_time - start_time);

	ppm_write(bitmap, DIM, DIM, fp);

	fclose(fp);
	free(bitmap);
	free(temp_s);
	hipFree(d_bitmap);
	hipFree(d_temp_s);

	cout << "CUDA ray tracing: " << duration.count() / 1000000.0 << " sec" << endl;
	cout << "[result.ppm] was generated." << endl;
	return 0;
}